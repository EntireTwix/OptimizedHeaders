#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

template <typename T>
__global__ void gpu_matrix_mult(T *a, T *b, T *c, uint16_t ax_sz, uint16_t ay_sz, uint16_t bx_sz, uint16_t by_sz)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < ay_sz && col < bx_sz)
    {
        for (uint16_t i = 0; i < ax_sz; ++i)
        {
            c[col + (row * bx_sz)] += a[i + (row * ax_sz)] * b[col + (i * bx_sz)];
        }
    }
}

template <typename T>
void mat_matrix_mult(T *a, T *b, T *c, uint16_t ax_sz, uint16_t ay_sz, uint16_t bx_sz, uint16_t by_sz, uint16_t cx_sz, uint16_t cy_sz)
{
    T *reg_a;
    T *reg_b;
    T *reg_c;

    hipMalloc((void **)&reg_a, ax_sz * ay_sz * sizeof(T));
    hipMalloc((void **)&reg_b, bx_sz * by_sz * sizeof(T));
    hipMalloc((void **)&reg_c, cx_sz * cy_sz * sizeof(T));

    hipMemcpy(reg_a, a, ax_sz * ay_sz * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(reg_b, b, bx_sz * by_sz * sizeof(T), hipMemcpyHostToDevice);

    unsigned int grid_rows = (ax_sz + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (by_sz + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(reg_a, reg_b, reg_c, ax_sz, ay_sz, bx_sz, by_sz);
    hipMemcpy(c, reg_c, sizeof(T) * cx_sz * cy_sz, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(reg_a);
    hipFree(reg_b);
    hipFree(reg_c);
}
